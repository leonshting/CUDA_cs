#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

typedef unsigned char u8;

typedef struct cell {
    u8 state;
    size_t* neighbor;
    u8 neighborSize;
} cell;

typedef enum type {life, koNeiman, koNeimanMur, koMur} type;

u8*** hStates;
size_t hX, hY, hZ;
type hT;

__device__ u8* dStates;
__device__ size_t *pdX, *pdY, *pdZ;
__device__ type *pdT;

__device__ cell* dCurrent;
__device__ cell* dNext;
__device__ size_t* pdFullSize;

void readInput(const char* inputFile)
{
    FILE* input = fopen(inputFile, "r");
    if (input == NULL) {
        printf("Can't open file %s\n", inputFile);
        exit(-1);
    }
    
    u8 firstLine = 1;
    const int LINE_SIZE = 100;
    char line[LINE_SIZE];
    size_t x, y, z;
    u8 cellState;
    int typeNumber;
    
    size_t i;
    size_t wordStart;
    u8 inWord;
    u8 separator;
    char* word;
    size_t wordSize;
    size_t wordCount;
    
    while (fgets(line, LINE_SIZE, input)) {
        wordCount = 0;
        wordStart = 0;
        inWord = 0;
        i = 0;
        if (firstLine) {
            while (line[i] != '\0') {
                separator = (line[i] == ' ' || line[i] == '\n') ? 1 : 0;
                if (inWord) {
                    if (separator) {
                        inWord = 0;
                        wordSize = i - wordStart;
                        word = (char*) malloc(wordSize + 1);
                        memcpy(word, line + wordStart, (i - wordStart) * sizeof(char));
                        word[wordSize] = '\0';
                        
                        switch (wordCount) {
                            case 0:
                                hX = (size_t)atoi(word);
                                break;
                            case 1:
                                hY = (size_t)atoi(word);
                                break;
                            case 2:
                                hZ = (size_t)atoi(word);
                                break;
                            case 3:
                                typeNumber = atoi(word);
                                switch (typeNumber) {
                                    case 1:
                                        hT = life;
                                        break;
                                    case 2:
                                        hT = koNeiman;
                                        break;
                                    case 3:
                                        hT = koNeimanMur;
                                        break;
                                    case 4:
                                        hT = koMur;
                                        break;
                                    default:
                                        printf("Wrong type of simulation: %d\n", typeNumber);
                                        exit(-1);
                                }
                                break;
                            default:
                                printf("Too much words in a line: %s\n", line);
                                exit(-1);
                        }
                        
                        free(word);
                        wordCount++;
                    }
                }
                else if (!separator) {
                    inWord = 1;
                    wordStart = i;
                }
                i++;
            }
            firstLine = 0;
            
            hStates = (u8***) malloc(hX * sizeof(u8**));
            for (x = 0; x < hX; x++) {
                hStates[x] = (u8**) malloc(hY * sizeof(u8*));
                for (y = 0; y < hY; y++) {
                    hStates[x][y] = (u8*) malloc(hZ * sizeof(u8));
                    for (z = 0; z < hZ; z++)
                        hStates[x][y][z] = 0;
                }
            }
        }
        else {
            while (line[i] != '\0') {
                separator = (line[i] == ' ' || line[i] == '\n') ? 1 : 0;
                if (inWord) {
                    if (separator) {
                        inWord = 0;
                        wordSize = i - wordStart;
                        word = (char*) malloc(wordSize + 1);
                        memcpy(word, line + wordStart, (i - wordStart) * sizeof(char));
                        word[wordSize] = '\0';
                        
                        switch (wordCount) {
                            case 0:
                                cellState = (u8)atoi(word);
                                if (cellState == 0)
                                    goto stop;
                                break;
                            case 1:
                                x = (size_t)atoi(word);
                                break;
                            case 2:
                                y = (size_t)atoi(word);
                                break;
                            case 3:
                                z = (size_t)atoi(word);
                                break;
                            default:
                                printf("Too much words in a line: %s\n", line);
                                exit(-1);
                        }
                        free(word);
                        wordCount++;
                    }
                }
                else if (!separator) {
                    inWord = 1;
                    wordStart = i;
                }
                i++;
            }
            hStates[x][y][z] = cellState;
        }
    }
    
stop:
    fclose(input);
}

void passStatesToDevice(u8 ***hStates, type *hT, size_t *hX, size_t *hY, size_t *hZ, )
{
    size_t i, j;
    hipMalloc((void**)&dStates, hX * hY * hZ * sizeof(u8));
    for (i = 0; i < hX; i++)
        for (j = 0; j < hY; j++)
            hipMemcpy(&dStates[i * hY * hZ + j * hZ], hStates[i][j], hZ * sizeof(u8), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pdX, sizeof(size_t));
    hipMemcpy(pdX, &hX, sizeof(size_t), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pdY, sizeof(size_t));
    hipMemcpy(pdY, &hY, sizeof(size_t), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pdZ, sizeof(size_t));
    hipMemcpy(pdZ, &hZ, sizeof(size_t), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pdT, sizeof(type));
    hipMemcpy(pdT, &hT, sizeof(type), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&pdFullSize, sizeof(size_t));
    size_t size = hX * hY * hZ;
    hipMemcpy(pdFullSize, &size, sizeof(size_t), hipMemcpyHostToDevice);
    
    hipMalloc((void**)&dCurrent, size * sizeof(cell));
    hipMalloc((void**)&dNext, size * sizeof(cell));
    
    u8 neighborSize = 6;
    for (i = 0; i < size; i++) {
        hipMemcpy(&dCurrent[i].neighborSize, &neighborSize, sizeof(u8), hipMemcpyHostToDevice);
        hipMemcpy(&dNext[i].neighborSize, &neighborSize, sizeof(u8), hipMemcpyHostToDevice);
                
        size_t* tmp1;
        size_t* tmp2;
        hipMalloc((void**)&tmp1, neighborSize * sizeof(size_t));
        hipMalloc((void**)&tmp2, neighborSize * sizeof(size_t));
        hipMemcpy(&dCurrent[i].neighbor, &tmp1, neighborSize * sizeof(size_t), hipMemcpyDeviceToDevice);
        hipMemcpy(&dNext[i].neighbor, &tmp2, neighborSize * sizeof(size_t), hipMemcpyDeviceToDevice);
    }
}

__device__ void idx3to1(size_t x, size_t y, size_t z, size_t* i)
{
    *i = x * (*pdY) * (*pdZ) + y * (*pdZ) + z;
}

__device__ void idx1to3(size_t i, size_t* x, size_t* y, size_t* z)
{
    *x = i / (*pdY) / (*pdZ);
    i -= (*x) * (*pdY) * (*pdZ);
    *y = i / (*pdZ);
    i -= (*y) * (*pdZ);
    *z = i;
}

__device__ size_t plus(size_t i, size_t* max)
{
    return (i == *max - 1) ? 0 : ++i;
}

__device__ size_t minus(size_t i, size_t* max)
{
    return (i == 0) ? *max - 1 : --i;
}

__global__ void transformStatesIntoCells()
{
    int idx = threadIdx.x;
    while (idx <= *pdFullSize) {
        dCurrent[idx].state = dNext[idx].state = dStates[idx];
        
        size_t x, y, z;
        idx1to3(idx, &x, &y, &z);
        
        size_t xn[2], yn[2], zn[2];
        xn[0] = minus(x, pdX);
        xn[1] = plus(x, pdX);
        yn[0] = minus(y, pdY);
        yn[1] = plus(y, pdY);
        zn[0] = minus(z, pdZ);
        zn[1] = plus(z, pdZ);
        
        int i, j, k;
        size_t neighborIdx;
        int neighborCount = 0;
        for (i = 0; i < 2; i++)
            for (j = 0; j < 2; j++)
                for (k = 0; k < 2; k++) {
                    idx3to1(xn[i], yn[j], zn[k], &neighborIdx);
                    dCurrent[idx].neighbor[neighborCount] = dNext[idx].neighbor[neighborCount] = neighborIdx;
                    neighborCount++;
                }
        
        idx += blockDim.x;
    }
    __syncthreads();
}

__global__ void calc()
{
    int idx = threadIdx.x;
    while (idx <= *pdFullSize) {
        u8 s = 0;
        int i;
        for (i = 0; i < dCurrent[idx].neighborSize; i++)
            if (dCurrent[dCurrent[idx].neighbor[i]].state)
                s++;
        
        if (dCurrent[idx].state) {
            if (s < 4)
                dNext[idx].state = 0;
        }
        else {
            if (s >= 6)
                dNext[idx].state = 1;
        }
        idx += blockDim.x;
    }
    __syncthreads();
    
    idx = threadIdx.x;
    while (idx <= *pdFullSize) {
        dCurrent[idx].state = dNext[idx].state;
        idx += blockDim.x;
    }
    __syncthreads();
}

__global__ void transformCellsIntoStates()
{
    int idx = threadIdx.x;
    while (idx <= *pdFullSize) {
        dStates[idx] = dCurrent[idx].state;
        idx += blockDim.x;
    }
    __syncthreads();
}

void getDataFromDevice(size_t nThreads)
{
    transformCellsIntoStates<<<1, nThreads>>>();
    
    size_t i, j;
    for (i = 0; i < hX; i++)
        for (j = 0; j < hY; j++)
            hipMemcpy(hStates[i][j], &dStates[i * hY * hZ + j * hZ], hZ * sizeof(u8), hipMemcpyDeviceToHost);
}

void print(const char* outputFile)
{
    FILE* output = fopen(outputFile, "a");
    if (output == NULL) {
        printf("Can't open file %s\n", outputFile);
        exit(-1);
    }
    
    size_t i, j, k;
    for (i = 0; i < hX; i++)
        for (j = 0; j < hY; j++)
            for (k = 0; k < hZ; k++)
                fprintf(output, "%d %ld %ld %ld\n", hStates[i][j][k], i, j, k);
    fprintf(output, "0 0 0 0\n");
    fclose(output);
}

void printResults(const char* outputFile, size_t nThreads)
{
    getDataFromDevice(nThreads);
    print(outputFile);
}

void clean()
{
    hipFree(dStates);
    hipFree(pdX);
    hipFree(pdY);
    hipFree(pdZ);
    hipFree(pdT);
    
    /*size_t i;
    for (i = 0; i < hX * hY * hZ; i++) {
        hipFree(dCurrent[i].neighbor);
        hipFree(dNext[i].neighbor);
    }*/
    hipFree(dCurrent);
    hipFree(dNext);
    hipFree(pdFullSize);
    
    size_t x, y;
    for (x = 0; x < hX; x++) {
        for (y = 0; y < hY; y++)
            free(hStates[x][y]);
        free(hStates[x]);
    }
    hipFree(hStates);
}

void gameOfLife(const char* inputFile, int nSteps, int outputInterval, const char* outputFile)
{
    readInput(inputFile);
    printf("Input file has been read\n");
    passStatesToDevice();
    printf("States have been copied to device\n");
    
    int device;
    hipGetDevice(&device);
    
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    transformStatesIntoCells<<<1, prop.maxThreadsDim[0]>>>();
    printf("Neighbors have been set\n");
    
    FILE* output = fopen(outputFile, "w");
    if (output == NULL) {
        printf("Can't open file %s\n", outputFile);
        exit(-1);
    }
    fclose(output);
    
    print(outputFile);
    printf("Output\n");
    
    int i;
    for (i = 1; i <= nSteps; i++) {
        calc<<<1, prop.maxThreadsDim[0]>>>();
        printf("Step %d\n", i);
        if (i % outputInterval == 0) {
            printResults(outputFile, prop.maxThreadsDim[0]);
            printf("Output\n");
        }
    }
    
    clean();
    printf("Memory has been set free\n");
}

int main(int argc, const char * argv[])
{
    if (argc != 5)
        printf("Usage: %s inputFile nSteps outputInterval outputFile\n", argv[0]);
    else
    {
        const char* inputFile = argv[1];
        int nSteps = atoi(argv[2]);
        int outputInterval = atoi(argv[3]);
        const char* outputFile = argv[4];
        
        gameOfLife(inputFile, nSteps, outputInterval, outputFile);
    }
    return 0;
}
